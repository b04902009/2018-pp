
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCKSIZE 512
__global__ void prefix(const char *cuStr, int *cuPos, int strLen){
    int localIdx = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalIdx >= strLen) return;

    __shared__ int buf[BLOCKSIZE];
    buf[localIdx] = cuStr[globalIdx] == ' ' ? localIdx : -1;
    __syncthreads();

    int cnt = 0;
    while(buf[localIdx] < 0 && localIdx > cnt){
        cnt++;
        buf[localIdx] = buf[localIdx-cnt];
    }
    cuPos[globalIdx] = localIdx - buf[localIdx];
}
__global__ void combine(int *cuPos, int strLen){
    int localIdx = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalIdx >= strLen) return;

    if(blockIdx.x > 0 && cuPos[globalIdx] == localIdx + 1)
        cuPos[globalIdx] += cuPos[blockIdx.x * blockDim.x - 1];
}
void labeling(const char *cuStr, int *cuPos, int strLen){
    const int GRIDSIZE = (strLen+BLOCKSIZE) / BLOCKSIZE;
    prefix <<< GRIDSIZE, BLOCKSIZE >>> (cuStr, cuPos, strLen);
    combine <<< GRIDSIZE, BLOCKSIZE >>> (cuPos, strLen);
}